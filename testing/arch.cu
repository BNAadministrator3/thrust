#include <unittest/unittest.h>

#if defined(__HIPCC__)

#include <thrust/detail/backend/cuda/arch.h>

using namespace thrust::detail::backend::cuda::arch;

void set_compute_capability(hipDeviceProp_t& properties, int major, int minor)
{
  properties.major = major;
  properties.minor = minor;
}

void set_G80(hipDeviceProp_t& properties)
{
  set_compute_capability(properties, 1, 0);
  properties.multiProcessorCount         = 16;
  properties.sharedMemPerBlock           = 16384;
  properties.regsPerBlock                = 8192;
  properties.warpSize                    = 32;
  properties.maxThreadsPerBlock          = 512;
  properties.maxThreadsPerMultiProcessor = 768;
}

void set_G84(hipDeviceProp_t& properties)
{
  set_compute_capability(properties, 1, 1);
  properties.multiProcessorCount         = 4;
  properties.sharedMemPerBlock           = 16384;
  properties.regsPerBlock                = 8192;
  properties.warpSize                    = 32;
  properties.maxThreadsPerBlock          = 512;
  properties.maxThreadsPerMultiProcessor = 768;
}

void set_GT200(hipDeviceProp_t& properties)
{
  set_compute_capability(properties, 1, 3);
  properties.multiProcessorCount         = 30;
  properties.sharedMemPerBlock           = 16384;
  properties.regsPerBlock                = 16384;
  properties.warpSize                    = 32;
  properties.maxThreadsPerBlock          = 512;
  properties.maxThreadsPerMultiProcessor = 1024;
}

void set_func_attributes(hipFuncAttributes& attributes,
                         size_t constSizeBytes,           // Size of constant memory in bytes.
                         size_t localSizeBytes,           // Size of local memory in bytes.
                         int maxThreadsPerBlock,          // Maximum number of threads per block.
                         int numRegs,                     // Number of registers used.
                         size_t sharedSizeBytes)          // Size of shared memory in bytes.
{
    attributes.constSizeBytes     = constSizeBytes;
    attributes.localSizeBytes     = localSizeBytes;
    attributes.maxThreadsPerBlock = maxThreadsPerBlock; 
    attributes.numRegs            = numRegs;
    attributes.sharedSizeBytes    = sharedSizeBytes;
}

void TestComputeCapability(void)
{
    hipDeviceProp_t properties;

    set_compute_capability(properties, 1, 0);
    ASSERT_EQUAL(compute_capability(properties), 10);

    set_compute_capability(properties, 1, 1);
    ASSERT_EQUAL(compute_capability(properties), 11);
    
    set_compute_capability(properties, 1, 3);
    ASSERT_EQUAL(compute_capability(properties), 13);
    
    set_compute_capability(properties, 2, 0);
    ASSERT_EQUAL(compute_capability(properties), 20);
    
    set_compute_capability(properties, 2, 1);
    ASSERT_EQUAL(compute_capability(properties), 21);
}
DECLARE_UNITTEST(TestComputeCapability);


void TestMaxActiveThreads(void)
{
    hipDeviceProp_t properties;

    set_G80(properties);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 768);
    
    set_G84(properties);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 768);
    
    set_GT200(properties);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 1024);
}
DECLARE_UNITTEST(TestMaxActiveThreads);


void TestMaxActiveBlocks(void)
{
    hipDeviceProp_t properties;
    hipFuncAttributes attributes;

    // Kernel #1 : Full Occupancy on all devices
    set_func_attributes(attributes, 0, 0, 512, 10, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 3);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 3);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 4);
    
    // Kernel #2 : 2/3rds Occupancy on G8x and 100% on GT200
    set_func_attributes(attributes, 0, 0, 512, 16, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 4);
    
    // Kernel #3 : 1/3rds Occupancy on G8x and 75% on GT200
    set_func_attributes(attributes, 0, 0, 512, 20, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 3);
    
    // Kernel #4 : 1/3rds Occupancy on G8x and 50% on GT200
    set_func_attributes(attributes, 0, 0, 512, 21, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    
    // Kernel #5 : 2/3rds Occupancy on G8x and 50% on GT200
    set_func_attributes(attributes, 0, 0, 512, 10, 8192);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
}
DECLARE_UNITTEST(TestMaxActiveBlocks);


void TestMaxBlocksizeWithHighestOccupancy(void)
{
    hipDeviceProp_t properties;
    hipFuncAttributes attributes;
    
    // Kernel #1 : Full Occupancy on all devices
    set_func_attributes(attributes, 0, 0, 512, 10, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 384);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 512);
    
    // Kernel #2 : 2/3rds Occupancy on G8x and 100% on GT200
    set_func_attributes(attributes, 0, 0, 512, 16, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 512);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 512);
    
    // Kernel #3 : 50% Occupancy on G8x and 75% on GT200
    set_func_attributes(attributes, 0, 0, 256, 20, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 192);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 256);
    
    // Kernel #4 : 1/3rds Occupancy on G8x and 50% on GT200
    set_func_attributes(attributes, 0, 0, 384, 26, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 256);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 192);
    
    // Kernel #5 :100% Occupancy on G8x and GT200
    set_func_attributes(attributes, 0, 0, 512, 10, 8192);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 384);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize_with_highest_occupancy(properties, attributes), 512);
}
DECLARE_UNITTEST(TestMaxBlocksizeWithHighestOccupancy);


void TestMaxBlocksize(void)
{
    hipDeviceProp_t properties;
    hipFuncAttributes attributes;
    
    // Kernel #1 : Full Occupancy on all devices
    set_func_attributes(attributes, 0, 0, 512, 10, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
    
    // Kernel #2 : 2/3rds Occupancy on G8x and 100% on GT200
    set_func_attributes(attributes, 0, 0, 512, 16, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
    
    // Kernel #3 : 50% Occupancy on G8x and 75% on GT200
    set_func_attributes(attributes, 0, 0, 512, 20, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize(properties, attributes), 384);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
    
    // Kernel #4 : 1/3rds Occupancy on G8x and 50% on GT200
    set_func_attributes(attributes, 0, 0, 384, 26, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize(properties, attributes), 256);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize(properties, attributes), 384);
    
    // Kernel #5 :100% Occupancy on G8x and GT200
    set_func_attributes(attributes, 0, 0, 512, 10, 8192);
    
    set_G80(properties);   ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
    set_GT200(properties); ASSERT_EQUAL(max_blocksize(properties, attributes), 512);
}
DECLARE_UNITTEST(TestMaxBlocksize);

#endif // defined(__HIPCC__)

