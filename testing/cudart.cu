#include <unittest/unittest.h>
#include <hip/hip_runtime_api.h>
#include <thrust/detail/util/align.h>

void TestCudaMemcpyD2DNullPointerError(void)
{
  hipError_t result1 = hipMemcpy((void*)0, (void*)0, 1, hipMemcpyDeviceToDevice);
  hipError_t result2 = hipGetLastError();

  ASSERT_EQUAL(hipErrorInvalidValue, result1);
  ASSERT_EQUAL(hipErrorInvalidValue, result2);
}
DECLARE_UNITTEST(TestCudaMemcpyD2DNullPointerError);

template<typename T>
void TestCudaMallocResultAligned(const std::size_t n)
{
  T *ptr = 0;
  hipMalloc(&ptr, n * sizeof(T));
  hipFree(ptr);

  ASSERT_EQUAL(true, thrust::detail::util::is_aligned(ptr));
}
DECLARE_VARIABLE_UNITTEST(TestCudaMallocResultAligned);

